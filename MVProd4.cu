#include "hip/hip_runtime.h"
/* 4-GPU implementation for computing matrix-vector products in BEM, for lossy surface materials.
Users do not need to read or understand any method in this file. 
PLEASE DO NOT MODIFY THE FOLLOWING CODE. */

#include "MVProd4.h"

/// @brief 4-GPU implementation for computing matrix-vector products in BEM, for lossy surface materials
/// @param est: an Estimate object with information on the simulated surface
/// @param singular: a Singular object that computes the matrix elements with singularties in the underlying integrals
/// @param grid: a Grid object with information on the 3D grid of point sources
/// @param ind0, ind1, ind2, ind3: indices of the GPUs used
MVProd4::MVProd4(Estimate* est, Singular* singular, Grid* grid, int ind0, int ind1, int ind2, int ind3): MVProd(est, singular, grid) {
    alpha.x = 1.0f;
    alpha.y = 0.0f;
    beta.x = 1.0f;
    beta.y = 0.0f;
    h_y1 = VectorXcf::Zero(hori_num);
    h_y2 = VectorXcf::Zero(vert_num);
    h_y3 = VectorXcf::Zero(hori_num);
    h_y4 = VectorXcf::Zero(vert_num);
    devNumber[0] = ind0;
    devNumber[1] = ind1;
    devNumber[2] = ind2;
    devNumber[3] = ind3;
    hipsparseCreate(&handle0);
    hipsparseCreate(&handle1);
    hipsparseCreate(&handle2);
    hipsparseCreate(&handle3);
    initializeNear();
    initializeFar();
}

/// @brief Allocate sparse correction matrices and the data structures holding the input, output vectors on the GPU
void MVProd4::initializeNear() {
    // Sparsity patterns are stored in binary files and transferred to GPUs
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        nnzA[dev] = est->A[dev].rows();
        hipMalloc(&d_Arows[dev], nnzA[dev] * sizeof(int));
        hipMemcpy(d_Arows[dev], (int*)((est->A[dev].block(0, 0, nnzA[dev], 1)).data()), nnzA[dev] * sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_Acols[dev], nnzA[dev] * sizeof(int));
        hipMemcpy(d_Acols[dev], (int*)((est->A[dev].block(0, 1, nnzA[dev], 1)).data()), nnzA[dev] * sizeof(int), hipMemcpyHostToDevice);
        nnzB[dev] = est->B[dev].rows();
        hipMalloc(&d_Brows[dev], nnzB[dev] * sizeof(int));
        hipMemcpy(d_Brows[dev], (int*)((est->B[dev].block(0, 0, nnzB[dev], 1)).data()), nnzB[dev] * sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_Bcols[dev], nnzB[dev] * sizeof(int));
        hipMemcpy(d_Bcols[dev], (int*)((est->B[dev].block(0, 1, nnzB[dev], 1)).data()), nnzB[dev] * sizeof(int), hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();

    // Create sparse matrices and dense vectors on GPUs
    Vector4i numrows(hori_num, hori_num, hori_num, vert_num);
    Vector4i numcols(hori_num, vert_num, vert_num, vert_num);
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMalloc(&d_Aee[dev], nnzA[dev] * sizeof(hipComplex));
        hipMalloc(&d_Aem[dev], nnzA[dev] * sizeof(hipComplex));
        hipMalloc(&d_Amm[dev], nnzA[dev] * sizeof(hipComplex));
        hipMalloc(&d_Bee[dev], nnzB[dev] * sizeof(hipComplex));
        hipMalloc(&d_Bem[dev], nnzB[dev] * sizeof(hipComplex));
        hipMalloc(&d_Bmm[dev], nnzB[dev] * sizeof(hipComplex));
        hipsparseCreateCoo(&Aee[dev], numrows(dev), numcols(dev), nnzA[dev], d_Arows[dev], d_Acols[dev], d_Aee[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipsparseCreateCoo(&Aem[dev], numrows(dev), numcols(dev), nnzA[dev], d_Arows[dev], d_Acols[dev], d_Aem[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipsparseCreateCoo(&Amm[dev], numrows(dev), numcols(dev), nnzA[dev], d_Arows[dev], d_Acols[dev], d_Amm[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipsparseCreateCoo(&Bee[dev], numrows(dev), numcols(dev), nnzB[dev], d_Brows[dev], d_Bcols[dev], d_Bee[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipsparseCreateCoo(&Bem[dev], numrows(dev), numcols(dev), nnzB[dev], d_Brows[dev], d_Bcols[dev], d_Bem[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipsparseCreateCoo(&Bmm[dev], numrows(dev), numcols(dev), nnzB[dev], d_Brows[dev], d_Bcols[dev], d_Bmm[dev], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
        hipMalloc(&d_x1[dev], hori_num * sizeof(hipComplex));
        hipMalloc(&d_x2[dev], vert_num * sizeof(hipComplex));
        hipMalloc(&d_x3[dev], hori_num * sizeof(hipComplex));
        hipMalloc(&d_x4[dev], vert_num * sizeof(hipComplex));
        hipMalloc(&d_y1[dev], hori_num * sizeof(hipComplex));
        hipMalloc(&d_y2[dev], vert_num * sizeof(hipComplex));
        hipMalloc(&d_y3[dev], hori_num * sizeof(hipComplex));
        hipMalloc(&d_y4[dev], vert_num * sizeof(hipComplex));
        hipsparseCreateDnVec(&x1[dev], hori_num, d_x1[dev], HIP_C_32F);
        hipsparseCreateDnVec(&x2[dev], vert_num, d_x2[dev], HIP_C_32F);
        hipsparseCreateDnVec(&x3[dev], hori_num, d_x3[dev], HIP_C_32F);
        hipsparseCreateDnVec(&x4[dev], vert_num, d_x4[dev], HIP_C_32F);
        hipsparseCreateDnVec(&y1[dev], hori_num, d_y1[dev], HIP_C_32F);
        hipsparseCreateDnVec(&y2[dev], vert_num, d_y2[dev], HIP_C_32F);
        hipsparseCreateDnVec(&y3[dev], hori_num, d_y3[dev], HIP_C_32F);
        hipsparseCreateDnVec(&y4[dev], vert_num, d_y4[dev], HIP_C_32F);
        hipMalloc(&d_work[dev], 0);
    }
    hipDeviceSynchronize();

    // Transfer matrix element data that require singularity removal computation to GPUs
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        VectorXcf matrix_data;
        singular->computeQuarter(dev);
        matrix_data = singular->quarter.block(0, 0, nnzA[dev], 1);
        hipMalloc(&d_base1[dev], nnzA[dev] * sizeof(hipComplex));
        hipMemcpy(d_base1[dev], (fcomp*)(matrix_data.data()), nnzA[dev] * sizeof(hipComplex), hipMemcpyHostToDevice);
        matrix_data = singular->quarter.block(0, 1, nnzA[dev], 1);
        hipMalloc(&d_base2[dev], nnzA[dev] * sizeof(hipComplex));
        hipMemcpy(d_base2[dev], (fcomp*)(matrix_data.data()), nnzA[dev] * sizeof(hipComplex), hipMemcpyHostToDevice);
        matrix_data = singular->quarter.block(0, 2, nnzA[dev], 1);
        hipMemcpy(d_Aem[dev], (fcomp*)(matrix_data.data()), nnzA[dev] * sizeof(hipComplex), hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();

    // Allocate height field data and necessary quadrature points / weights
    MatrixXf zdata = est->zvals.cast<float>();
    VectorXf xhigh = quadrature_points.block(3, 0, 1, 4).transpose().cast<float>();
    VectorXf whigh = quadrature_weights.block(3, 0, 1, 4).transpose().cast<float>();
    VectorXf xlow = quadrature_points.block(1, 0, 1, 2).transpose().cast<float>();
    VectorXf wlow = quadrature_weights.block(1, 0, 1, 2).transpose().cast<float>();
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMalloc(&zvals[dev], (Nx + 1) * (Ny + 1) * sizeof(float));
        hipMemcpy(zvals[dev], (float*)(zdata.data()), (Nx + 1) * (Ny + 1) * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&xvech[dev], 4 * sizeof(float));
        hipMemcpy(xvech[dev], (float*)(xhigh.data()), 4 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&wvech[dev], 4 * sizeof(float));
        hipMemcpy(wvech[dev], (float*)(whigh.data()), 4 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&xvecl[dev], 2 * sizeof(float));
        hipMemcpy(xvecl[dev], (float*)(xlow.data()), 2 * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&wvecl[dev], 2 * sizeof(float));
        hipMemcpy(wvecl[dev], (float*)(wlow.data()), 2 * sizeof(float), hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();
}

/// @brief Allocate memory for point source approximation coefficients on the GPU; create FFT plans for computation
void MVProd4::initializeFar() {
    // Allocate point source approximation coefficients and indices
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMalloc(&d_hori_x[dev], num_pts * hori_num * sizeof(Tfcomp));
        hipMalloc(&d_hori_z[dev], num_pts * hori_num * sizeof(Tfcomp));
        hipMalloc(&d_hori_d[dev], num_pts * hori_num * sizeof(Tfcomp));
        hipMalloc(&d_hori_f[dev], hori_row * hori_col * sizeof(int));
        hipMalloc(&d_hori_b[dev], num_pts * hori_num * sizeof(int));
        hipMalloc(&d_vert_y[dev], num_pts * vert_num * sizeof(Tfcomp));
        hipMalloc(&d_vert_z[dev], num_pts * vert_num * sizeof(Tfcomp));
        hipMalloc(&d_vert_d[dev], num_pts * vert_num * sizeof(Tfcomp));
        hipMalloc(&d_vert_f[dev], vert_row * vert_col * sizeof(int));
        hipMalloc(&d_vert_b[dev], num_pts * vert_num * sizeof(int));
    }
    hipDeviceSynchronize();

    // Allocate far matrix multiplication data structures
    for (int dev = 0; dev < 2; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMalloc(&g0_data[dev], N * sizeof(Tfcomp));
        hipMalloc(&g1_data[dev], N * sizeof(Tfcomp));
        hipMalloc(&geo0_data[dev], N * sizeof(Tfcomp));
        hipfftCreate(&plan[dev]);
        size_t* worksize = (size_t *)malloc(sizeof(size_t));
        hipfftMakePlan3d(plan[dev], totalX, totalY, totalZ, HIPFFT_C2C, worksize);
    }
    for (int dev = 2; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMalloc(&g2_data[dev - 2], N * sizeof(Tfcomp));
        hipMalloc(&g3_data[dev - 2], N * sizeof(Tfcomp));
        hipMalloc(&g4_data[dev - 2], N * sizeof(Tfcomp));
        hipMalloc(&geo1_data[dev - 2], N * sizeof(Tfcomp));
        hipMalloc(&geo2_data[dev - 2], N * sizeof(Tfcomp));
        hipfftCreate(&plan[dev]);
        size_t* worksize = (size_t *)malloc(sizeof(size_t));
        hipfftMakePlan3d(plan[dev], totalX, totalY, totalZ, HIPFFT_C2C, worksize);
    }
    hipDeviceSynchronize();
}

/// @brief Perform initializations for computing matrix-vector products in a simulation with given media parameters and wavelengths
/// @brief Compute the point source approximations, the sparse correction matrices, and Fourier transform of Green's functions
/// @param eta1: index of refraction of the medium where the light is incident from, usually 1.0 (air)
/// @param eta2: index of refraction of the surface material (could be complex-valued)
/// @param lambda: the currently simulated wavelength
void MVProd4::setParameters(double eta1, dcomp eta2, double lambda) {
    // Initialize parameters
    double omega = c / lambda * 2 * M_PI;
    dcomp eps1 = 1 / (mu * c * c) * eta1 * eta1;
    dcomp eps2 = 1 / (mu * c * c) * eta2 * eta2;
    Tfcomp e1 = Tfcomp((float)eta1, 0.0f);
    Tfcomp e2 = Tfcomp((float)real(eta2), (float)imag(eta2));
    Tfcomp k1 = (float)(2 * M_PI / lambda) * e1;
    Tfcomp k2 = (float)(2 * M_PI / lambda) * e2;
    Tfcomp const1 = Tfcomp((float)real(cuDB * omega * mu), (float)imag(cuDB * omega * mu));
    Tfcomp const21 = Tfcomp((float)real(cuDB / (omega * eps1)), (float)imag(cuDB / (omega * eps1)));
    Tfcomp const22 = Tfcomp((float)real(cuDB / (omega * eps2)), (float)imag(cuDB / (omega * eps2)));
    Tfcomp c1 = 2.0f * const1;
    Tfcomp c2 = const21 + const22;
    Tfcomp c3 = -(e1 * e1 + e2 * e2) * const1;
    Tfcomp c4 = -e1 * e1 * const21 - e2 * e2 * const22;
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        updateEEMM<<< nnzA[dev] / 256 + 1, 256 >>>(nnzA[dev], d_Aee[dev], d_Amm[dev], d_base1[dev], d_base2[dev], c1, c2, c3, c4);
    }
    hipDeviceSynchronize();

    // Compute point source approximation coefficients and transfer to GPUs
    grid->computeCoefficients(eta1, eta2, lambda);
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMemcpy(d_hori_x[dev], (fcomp*)(grid->hori_x.data()), num_pts * hori_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_hori_z[dev], (fcomp*)(grid->hori_z.data()), num_pts * hori_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_hori_d[dev], (fcomp*)(grid->hori_d.data()), num_pts * hori_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_hori_f[dev], (int*)(grid->hori_f.data()), hori_row * hori_col * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_hori_b[dev], (int*)(grid->hori_b.data()), num_pts * hori_num * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_vert_y[dev], (fcomp*)(grid->vert_y.data()), num_pts * vert_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_vert_z[dev], (fcomp*)(grid->vert_z.data()), num_pts * vert_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_vert_d[dev], (fcomp*)(grid->vert_d.data()), num_pts * vert_num * sizeof(fcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_vert_f[dev], (int*)(grid->vert_f.data()), vert_row * vert_col * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_vert_b[dev], (int*)(grid->vert_b.data()), num_pts * vert_num * sizeof(int), hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();

    // Compute near matrix elements
    hipSetDevice(devNumber[0]);
    clearData<<< nnzB[0] / 256 + 1, 256 >>>(d_Bee[0], nnzB[0]);
    clearData<<< nnzB[0] / 256 + 1, 256 >>>(d_Bem[0], nnzB[0]);
    clearData<<< nnzB[0] / 256 + 1, 256 >>>(d_Bmm[0], nnzB[0]);
    individual<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], Nx, Ny, zvals[0], d, k1, k2, const1, const21, const22, xvech[0], wvech[0], xvecl[0], wvecl[0], d_Brows[0], d_Bcols[0], 0, 0, 0, 0, 1, -1, -1, d_Bee[0], d_Bmm[0], d_Bem[0]);
    individual<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], Nx, Ny, zvals[0], d, k1, k2, const1, const21, const22, xvech[0], wvech[0], xvecl[0], wvecl[0], d_Brows[0], d_Bcols[0], 0, 0, 1, 0, 1, -1, 1, d_Bee[0], d_Bmm[0], d_Bem[0]);
    individual<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], Nx, Ny, zvals[0], d, k1, k2, const1, const21, const22, xvech[0], wvech[0], xvecl[0], wvecl[0], d_Brows[0], d_Bcols[0], 1, 0, 0, 0, 1, 1, -1, d_Bee[0], d_Bmm[0], d_Bem[0]);
    individual<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], Nx, Ny, zvals[0], d, k1, k2, const1, const21, const22, xvech[0], wvech[0], xvecl[0], wvecl[0], d_Brows[0], d_Bcols[0], 1, 0, 1, 0, 1, 1, 1, d_Bee[0], d_Bmm[0], d_Bem[0]);
    correctionHH<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], dx, dy, dz, k1, const1, const21, totalY, totalZ, d_Brows[0], d_Bcols[0], num_pts, d_hori_b[0], d_hori_x[0], d_hori_z[0], d_hori_d[0], d_Bee[0], d_Bem[0]);
    postProcess<<< nnzB[0] / 256 + 1, 256 >>>(nnzB[0], true, eta0FL, eta1, eta2, d_Brows[0], d_Bcols[0], d_Bee[0], d_Bmm[0], d_Bem[0]);
    hipSetDevice(devNumber[1]);
    clearData<<< nnzB[1] / 256 + 1, 256 >>>(d_Bee[1], nnzB[1]);
    clearData<<< nnzB[1] / 256 + 1, 256 >>>(d_Bem[1], nnzB[1]);
    clearData<<< nnzB[1] / 256 + 1, 256 >>>(d_Bmm[1], nnzB[1]);
    individual<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], Nx, Ny, zvals[1], d, k1, k2, const1, const21, const22, xvech[1], wvech[1], xvecl[1], wvecl[1], d_Brows[1], d_Bcols[1], 0, 0, 0, 0, 2, -1, -1, d_Bee[1], d_Bmm[1], d_Bem[1]);
    individual<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], Nx, Ny, zvals[1], d, k1, k2, const1, const21, const22, xvech[1], wvech[1], xvecl[1], wvecl[1], d_Brows[1], d_Bcols[1], 0, 0, 0, 1, 2, -1, 1, d_Bee[1], d_Bmm[1], d_Bem[1]);
    individual<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], Nx, Ny, zvals[1], d, k1, k2, const1, const21, const22, xvech[1], wvech[1], xvecl[1], wvecl[1], d_Brows[1], d_Bcols[1], 1, 0, 0, 0, 2, 1, -1, d_Bee[1], d_Bmm[1], d_Bem[1]);
    individual<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], Nx, Ny, zvals[1], d, k1, k2, const1, const21, const22, xvech[1], wvech[1], xvecl[1], wvecl[1], d_Brows[1], d_Bcols[1], 1, 0, 0, 1, 2, 1, 1, d_Bee[1], d_Bmm[1], d_Bem[1]);
    correctionHV<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], dx, dy, dz, k1, const1, const21, totalY, totalZ, d_Brows[1], d_Bcols[1], num_pts, d_hori_b[1], d_vert_b[1], d_hori_x[1], d_hori_z[1], d_hori_d[1], d_vert_y[1], d_vert_z[1], d_vert_d[1], d_Bee[1], d_Bem[1]);
    postProcess<<< nnzB[1] / 256 + 1, 256 >>>(nnzB[1], false, eta0FL, eta1, eta2, d_Brows[1], d_Bcols[1], d_Bee[1], d_Bmm[1], d_Bem[1]);
    hipSetDevice(devNumber[2]);
    clearData<<< nnzB[2] / 256 + 1, 256 >>>(d_Bee[2], nnzB[2]);
    clearData<<< nnzB[2] / 256 + 1, 256 >>>(d_Bem[2], nnzB[2]);
    clearData<<< nnzB[2] / 256 + 1, 256 >>>(d_Bmm[2], nnzB[2]);
    individual<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], Nx, Ny, zvals[2], d, k1, k2, const1, const21, const22, xvech[2], wvech[2], xvecl[2], wvecl[2], d_Brows[2], d_Bcols[2], 0, 0, 0, 0, 2, -1, -1, d_Bee[2], d_Bmm[2], d_Bem[2]);
    individual<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], Nx, Ny, zvals[2], d, k1, k2, const1, const21, const22, xvech[2], wvech[2], xvecl[2], wvecl[2], d_Brows[2], d_Bcols[2], 0, 0, 0, 1, 2, -1, 1, d_Bee[2], d_Bmm[2], d_Bem[2]);
    individual<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], Nx, Ny, zvals[2], d, k1, k2, const1, const21, const22, xvech[2], wvech[2], xvecl[2], wvecl[2], d_Brows[2], d_Bcols[2], 1, 0, 0, 0, 2, 1, -1, d_Bee[2], d_Bmm[2], d_Bem[2]);
    individual<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], Nx, Ny, zvals[2], d, k1, k2, const1, const21, const22, xvech[2], wvech[2], xvecl[2], wvecl[2], d_Brows[2], d_Bcols[2], 1, 0, 0, 1, 2, 1, 1, d_Bee[2], d_Bmm[2], d_Bem[2]);
    correctionHV<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], dx, dy, dz, k1, const1, const21, totalY, totalZ, d_Brows[2], d_Bcols[2], num_pts, d_hori_b[2], d_vert_b[2], d_hori_x[2], d_hori_z[2], d_hori_d[2], d_vert_y[2], d_vert_z[2], d_vert_d[2], d_Bee[2], d_Bem[2]);
    postProcess<<< nnzB[2] / 256 + 1, 256 >>>(nnzB[2], false, eta0FL, eta1, eta2, d_Brows[2], d_Bcols[2], d_Bee[2], d_Bmm[2], d_Bem[2]);
    hipSetDevice(devNumber[3]);
    clearData<<< nnzB[3] / 256 + 1, 256 >>>(d_Bee[3], nnzB[3]);
    clearData<<< nnzB[3] / 256 + 1, 256 >>>(d_Bem[3], nnzB[3]);
    clearData<<< nnzB[3] / 256 + 1, 256 >>>(d_Bmm[3], nnzB[3]);
    individual<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], Nx, Ny, zvals[3], d, k1, k2, const1, const21, const22, xvech[3], wvech[3], xvecl[3], wvecl[3], d_Brows[3], d_Bcols[3], 0, 0, 0, 0, 3, -1, -1, d_Bee[3], d_Bmm[3], d_Bem[3]);
    individual<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], Nx, Ny, zvals[3], d, k1, k2, const1, const21, const22, xvech[3], wvech[3], xvecl[3], wvecl[3], d_Brows[3], d_Bcols[3], 0, 0, 0, 1, 3, -1, 1, d_Bee[3], d_Bmm[3], d_Bem[3]);
    individual<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], Nx, Ny, zvals[3], d, k1, k2, const1, const21, const22, xvech[3], wvech[3], xvecl[3], wvecl[3], d_Brows[3], d_Bcols[3], 0, 1, 0, 0, 3, 1, -1, d_Bee[3], d_Bmm[3], d_Bem[3]);
    individual<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], Nx, Ny, zvals[3], d, k1, k2, const1, const21, const22, xvech[3], wvech[3], xvecl[3], wvecl[3], d_Brows[3], d_Bcols[3], 0, 1, 0, 1, 3, 1, 1, d_Bee[3], d_Bmm[3], d_Bem[3]);
    correctionVV<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], dx, dy, dz, k1, const1, const21, totalY, totalZ, d_Brows[3], d_Bcols[3], num_pts, d_vert_b[3], d_vert_y[3], d_vert_z[3], d_vert_d[3], d_Bee[3], d_Bem[3]);
    postProcess<<< nnzB[3] / 256 + 1, 256 >>>(nnzB[3], true, eta0FL, eta1, eta2, d_Brows[3], d_Bcols[3], d_Bee[3], d_Bmm[3], d_Bem[3]);
    hipDeviceSynchronize();

    // Compute Green's function values and perform Fourier transforms
    hipSetDevice(devNumber[0]);
    computeGreens<<< N / 256 + 1, 256 >>>(g0_data[0], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 0);
    hipfftExecC2C(plan[0], (hipfftComplex*)g0_data[0], (hipfftComplex*)g0_data[0], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g1_data[0], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 1);
    hipfftExecC2C(plan[0], (hipfftComplex*)g1_data[0], (hipfftComplex*)g1_data[0], HIPFFT_FORWARD);
    hipSetDevice(devNumber[1]);
    computeGreens<<< N / 256 + 1, 256 >>>(g0_data[1], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 2);
    hipfftExecC2C(plan[1], (hipfftComplex*)g0_data[1], (hipfftComplex*)g0_data[1], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g1_data[1], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 3);
    hipfftExecC2C(plan[1], (hipfftComplex*)g1_data[1], (hipfftComplex*)g1_data[1], HIPFFT_FORWARD);
    hipSetDevice(devNumber[2]);
    computeGreens<<< N / 256 + 1, 256 >>>(g2_data[0], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 4);
    hipfftExecC2C(plan[2], (hipfftComplex*)g2_data[0], (hipfftComplex*)g2_data[0], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g3_data[0], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 5);
    hipfftExecC2C(plan[2], (hipfftComplex*)g3_data[0], (hipfftComplex*)g3_data[0], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g4_data[0], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 6);
    hipfftExecC2C(plan[2], (hipfftComplex*)g4_data[0], (hipfftComplex*)g4_data[0], HIPFFT_FORWARD);
    hipSetDevice(devNumber[3]);
    computeGreens<<< N / 256 + 1, 256 >>>(g2_data[1], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 4);
    hipfftExecC2C(plan[3], (hipfftComplex*)g2_data[1], (hipfftComplex*)g2_data[1], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g3_data[1], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 5);
    hipfftExecC2C(plan[3], (hipfftComplex*)g3_data[1], (hipfftComplex*)g3_data[1], HIPFFT_FORWARD);
    computeGreens<<< N / 256 + 1, 256 >>>(g4_data[1], dx, dy, dz, eta0FL, e1, k1, const1, const21, N, totalX, totalY, totalZ, 6);
    hipfftExecC2C(plan[3], (hipfftComplex*)g4_data[1], (hipfftComplex*)g4_data[1], HIPFFT_FORWARD);
    hipDeviceSynchronize();
}

/// @brief Perform matrix-vector multiplication using the BEM matrix
/// @param x: the input vector
/// @return The product vector
VectorXcf MVProd4::multiply(VectorXcf x) {
    VectorXcf h_x1 = x.block(0, 0, hori_num, 1);
    VectorXcf h_x2 = x.block(hori_num, 0, vert_num, 1);
    VectorXcf h_x3 = x.block(hori_num + vert_num, 0, hori_num, 1);
    VectorXcf h_x4 = x.block(2 * hori_num + vert_num, 0, vert_num, 1);
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMemcpy(d_x1[dev], (fcomp*)(h_x1.data()), hori_num * sizeof(Tfcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_x2[dev], (fcomp*)(h_x2.data()), vert_num * sizeof(Tfcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_x3[dev], (fcomp*)(h_x3.data()), hori_num * sizeof(Tfcomp), hipMemcpyHostToDevice);
        hipMemcpy(d_x4[dev], (fcomp*)(h_x4.data()), vert_num * sizeof(Tfcomp), hipMemcpyHostToDevice);
    }
    gpu0();
    gpu1();
    gpu2();
    gpu3();
    VectorXcf y = VectorXcf::Zero(2 * hori_num + 2 * vert_num);
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipMemcpy((fcomp*)(h_y1.data()), d_y1[dev], hori_num * sizeof(fcomp), hipMemcpyDeviceToHost);
        hipMemcpy((fcomp*)(h_y2.data()), d_y2[dev], vert_num * sizeof(fcomp), hipMemcpyDeviceToHost);
        hipMemcpy((fcomp*)(h_y3.data()), d_y3[dev], hori_num * sizeof(fcomp), hipMemcpyDeviceToHost);
        hipMemcpy((fcomp*)(h_y4.data()), d_y4[dev], vert_num * sizeof(fcomp), hipMemcpyDeviceToHost);
        y.block(0, 0, hori_num, 1) += h_y1;
        y.block(hori_num, 0, vert_num, 1) += h_y2;
        y.block(hori_num + vert_num, 0, hori_num, 1) += h_y3;
        y.block(2 * hori_num + vert_num, 0, vert_num, 1) += h_y4;
    }
    return y;
}

/// @brief Matrix-vector multiplication computations done on GPU 0
void MVProd4::gpu0() {
    // Initialization: clearing all vectors to become zero-valued
    hipSetDevice(devNumber[0]);
    hipsparseDnVecSetValues(x1[0], d_x1[0]);
    hipsparseDnVecSetValues(x2[0], d_x2[0]);
    hipsparseDnVecSetValues(x3[0], d_x3[0]);
    hipsparseDnVecSetValues(x4[0], d_x4[0]);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y1[0], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y2[0], vert_num);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y3[0], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y4[0], vert_num);
    hipsparseDnVecSetValues(y1[0], d_y1[0]);
    hipsparseDnVecSetValues(y2[0], d_y2[0]);
    hipsparseDnVecSetValues(y3[0], d_y3[0]);
    hipsparseDnVecSetValues(y4[0], d_y4[0]);

    // Sparse near matrix multiplication
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aee[0], x1[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aee[0], x1[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bee[0], x1[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bee[0], x1[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[0], x3[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[0], x3[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[0], x3[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[0], x3[0], &beta, y1[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[0], x1[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[0], x1[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[0], x1[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[0], x1[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Amm[0], x3[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Amm[0], x3[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bmm[0], x3[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);
    hipsparseSpMV(handle0, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bmm[0], x3[0], &beta, y3[0], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[0]);

    // Far matrix multiplication for block Zee: FFT pair 1
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[0], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[0], d_x1[0], hori_row, hori_col, num_pts, d_hori_x[0], d_hori_f[0]);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[0], geo0_data[0], N);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y1[0], geo0_data[0], hori_num, num_pts, N, d_hori_x[0], d_hori_b[0], false);
    
    // Far matrix multiplication for block Zee: FFT pair 2
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[0], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[0], d_x2[0], vert_row, vert_col, num_pts, d_vert_y[0], d_vert_f[0]);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[0], geo0_data[0], N);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_BACKWARD);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y2[0], geo0_data[0], vert_num, num_pts, N, d_vert_y[0], d_vert_b[0], false);
    
    // Far matrix multiplication for block Zee: FFT pair 3
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[0], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[0], d_x1[0], hori_row, hori_col, num_pts, d_hori_z[0], d_hori_f[0]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[0], d_x2[0], vert_row, vert_col, num_pts, d_vert_z[0], d_vert_f[0]);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[0], geo0_data[0], N);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y1[0], geo0_data[0], hori_num, num_pts, N, d_hori_z[0], d_hori_b[0], false);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y2[0], geo0_data[0], vert_num, num_pts, N, d_vert_z[0], d_vert_b[0], false);

    // Far matrix multiplication for block Zee: FFT pair 4
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[0], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[0], d_x1[0], hori_row, hori_col, num_pts, d_hori_d[0], d_hori_f[0]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[0], d_x2[0], vert_row, vert_col, num_pts, d_vert_d[0], d_vert_f[0]);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g1_data[0], geo0_data[0], N);
    hipfftExecC2C(plan[0], (hipfftComplex*)geo0_data[0], (hipfftComplex*)geo0_data[0], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y1[0], geo0_data[0], hori_num, num_pts, N, d_hori_d[0], d_hori_b[0], true);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y2[0], geo0_data[0], vert_num, num_pts, N, d_vert_d[0], d_vert_b[0], true);
}

/// @brief Matrix-vector multiplication computations done on GPU 1
void MVProd4::gpu1() {
    // Initialization: clearing all vectors to become zero-valued
    hipSetDevice(devNumber[1]);
    hipsparseDnVecSetValues(x1[1], d_x1[1]);
    hipsparseDnVecSetValues(x2[1], d_x2[1]);
    hipsparseDnVecSetValues(x3[1], d_x3[1]);
    hipsparseDnVecSetValues(x4[1], d_x4[1]);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y1[1], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y2[1], vert_num);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y3[1], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y4[1], vert_num);
    hipsparseDnVecSetValues(y1[1], d_y1[1]);
    hipsparseDnVecSetValues(y2[1], d_y2[1]);
    hipsparseDnVecSetValues(y3[1], d_y3[1]);
    hipsparseDnVecSetValues(y4[1], d_y4[1]);

    // Sparse near matrix multiplication
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aee[1], x2[1], &beta, y1[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bee[1], x2[1], &beta, y1[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[1], x4[1], &beta, y1[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[1], x4[1], &beta, y1[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aee[1], x1[1], &beta, y2[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bee[1], x1[1], &beta, y2[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[1], x3[1], &beta, y2[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[1], x3[1], &beta, y2[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[1], x2[1], &beta, y3[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[1], x2[1], &beta, y3[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Amm[1], x4[1], &beta, y3[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bmm[1], x4[1], &beta, y3[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[1], x1[1], &beta, y4[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[1], x1[1], &beta, y4[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Amm[1], x3[1], &beta, y4[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);
    hipsparseSpMV(handle1, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bmm[1], x3[1], &beta, y4[1], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[1]);

    // Far matrix multiplication for block Zmm: FFT pair 1
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[1], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[1], d_x3[1], hori_row, hori_col, num_pts, d_hori_x[1], d_hori_f[1]);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[1], geo0_data[1], N);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y3[1], geo0_data[1], hori_num, num_pts, N, d_hori_x[1], d_hori_b[1], true);
    
    // Far matrix multiplication for block Zmm: FFT pair 2
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[1], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[1], d_x4[1], vert_row, vert_col, num_pts, d_vert_y[1], d_vert_f[1]);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[1], geo0_data[1], N);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_BACKWARD);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y4[1], geo0_data[1], vert_num, num_pts, N, d_vert_y[1], d_vert_b[1], true);

    // Far matrix multiplication for block Zmm: FFT pair 3
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[1], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[1], d_x3[1], hori_row, hori_col, num_pts, d_hori_z[1], d_hori_f[1]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[1], d_x4[1], vert_row, vert_col, num_pts, d_vert_z[1], d_vert_f[1]);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g0_data[1], geo0_data[1], N);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y3[1], geo0_data[1], hori_num, num_pts, N, d_hori_z[1], d_hori_b[1], true);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y4[1], geo0_data[1], vert_num, num_pts, N, d_vert_z[1], d_vert_b[1], true);

    // Far matrix multiplication for block Zmm: FFT pair 4
    clearData<<< N / 256 + 1, 256 >>>(geo0_data[1], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo0_data[1], d_x3[1], hori_row, hori_col, num_pts, d_hori_d[1], d_hori_f[1]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo0_data[1], d_x4[1], vert_row, vert_col, num_pts, d_vert_d[1], d_vert_f[1]);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_FORWARD);
    convolve<<< N / 256 + 1, 256 >>>(g1_data[1], geo0_data[1], N);
    hipfftExecC2C(plan[1], (hipfftComplex*)geo0_data[1], (hipfftComplex*)geo0_data[1], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y3[1], geo0_data[1], hori_num, num_pts, N, d_hori_d[1], d_hori_b[1], false);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y4[1], geo0_data[1], vert_num, num_pts, N, d_vert_d[1], d_vert_b[1], false);
}

/// @brief Matrix-vector multiplication computations done on GPU 2
void MVProd4::gpu2() {
    // Initialization: clearing all vectors to become zero-valued
    hipSetDevice(devNumber[2]);
    hipsparseDnVecSetValues(x1[2], d_x1[2]);
    hipsparseDnVecSetValues(x2[2], d_x2[2]);
    hipsparseDnVecSetValues(x3[2], d_x3[2]);
    hipsparseDnVecSetValues(x4[2], d_x4[2]);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y1[2], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y2[2], vert_num);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y3[2], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y4[2], vert_num);
    hipsparseDnVecSetValues(y1[2], d_y1[2]);
    hipsparseDnVecSetValues(y2[2], d_y2[2]);
    hipsparseDnVecSetValues(y3[2], d_y3[2]);
    hipsparseDnVecSetValues(y4[2], d_y4[2]);

    // Sparse near matrix multiplication
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aee[2], x2[2], &beta, y1[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bee[2], x2[2], &beta, y1[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[2], x4[2], &beta, y1[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[2], x4[2], &beta, y1[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aee[2], x1[2], &beta, y2[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bee[2], x1[2], &beta, y2[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[2], x3[2], &beta, y2[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[2], x3[2], &beta, y2[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[2], x2[2], &beta, y3[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[2], x2[2], &beta, y3[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Amm[2], x4[2], &beta, y3[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bmm[2], x4[2], &beta, y3[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[2], x1[2], &beta, y4[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[2], x1[2], &beta, y4[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Amm[2], x3[2], &beta, y4[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);
    hipsparseSpMV(handle2, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bmm[2], x3[2], &beta, y4[2], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[2]);

    // Far matrix multiplication for block Zem: FFT group 1
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[0], N);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[0], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[0], d_x4[2], vert_row, vert_col, num_pts, d_vert_y[2], d_vert_f[2]);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo2_data[0], (hipfftComplex*)geo2_data[0], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g4_data[0], geo2_data[0], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[0], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo2_data[0], d_x3[2], hori_row, hori_col, num_pts, d_hori_z[2], d_hori_f[2]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[0], d_x4[2], vert_row, vert_col, num_pts, d_vert_z[2], d_vert_f[2]);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo2_data[0], (hipfftComplex*)geo2_data[0], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g3_data[0], geo2_data[0], N, true);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo1_data[0], (hipfftComplex*)geo1_data[0], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y1[2], geo1_data[0], hori_num, num_pts, N, d_hori_x[2], d_hori_b[2], false);

    // Far matrix multiplication for block Zem: FFT group 2
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[0], N);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g2_data[0], geo2_data[0], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[0], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo2_data[0], d_x3[2], hori_row, hori_col, num_pts, d_hori_x[2], d_hori_f[2]);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo2_data[0], (hipfftComplex*)geo2_data[0], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g4_data[0], geo2_data[0], N, true);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo1_data[0], (hipfftComplex*)geo1_data[0], HIPFFT_BACKWARD);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y2[2], geo1_data[0], vert_num, num_pts, N, d_vert_y[2], d_vert_b[2], false);

    // Far matrix multiplication for block Zem: FFT group 3
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[0], N);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g3_data[0], geo2_data[0], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[0], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[0], d_x4[2], vert_row, vert_col, num_pts, d_vert_y[2], d_vert_f[2]);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo2_data[0], (hipfftComplex*)geo2_data[0], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[0], g2_data[0], geo2_data[0], N, true);
    hipfftExecC2C(plan[2], (hipfftComplex*)geo1_data[0], (hipfftComplex*)geo1_data[0], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y1[2], geo1_data[0], hori_num, num_pts, N, d_hori_z[2], d_hori_b[2], false);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y2[2], geo1_data[0], vert_num, num_pts, N, d_vert_z[2], d_vert_b[2], false);
}

/// @brief Matrix-vector multiplication computations done on GPU 3
void MVProd4::gpu3() {
    // Initialization: clearing all vectors to become zero-valued
    hipSetDevice(devNumber[3]);
    hipsparseDnVecSetValues(x1[3], d_x1[3]);
    hipsparseDnVecSetValues(x2[3], d_x2[3]);
    hipsparseDnVecSetValues(x3[3], d_x3[3]);
    hipsparseDnVecSetValues(x4[3], d_x4[3]);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y1[3], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y2[3], vert_num);
    clearData<<< hori_num / 256 + 1, 256 >>>(d_y3[3], hori_num);
    clearData<<< vert_num / 256 + 1, 256 >>>(d_y4[3], vert_num);
    hipsparseDnVecSetValues(y1[3], d_y1[3]);
    hipsparseDnVecSetValues(y2[3], d_y2[3]);
    hipsparseDnVecSetValues(y3[3], d_y3[3]);
    hipsparseDnVecSetValues(y4[3], d_y4[3]);

    // Sparse near matrix multiplication
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aee[3], x2[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aee[3], x2[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bee[3], x2[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bee[3], x2[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[3], x4[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[3], x4[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[3], x4[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[3], x4[3], &beta, y2[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Aem[3], x2[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Aem[3], x2[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bem[3], x2[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bem[3], x2[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Amm[3], x4[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Amm[3], x4[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Bmm[3], x4[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);
    hipsparseSpMV(handle3, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, Bmm[3], x4[3], &beta, y4[3], HIP_C_32F, HIPSPARSE_SPMV_COO_ALG1, d_work[3]);

    // Far matrix multiplication for block Zme: FFT group 1
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[1], N);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[1], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[1], d_x2[3], vert_row, vert_col, num_pts, d_vert_y[3], d_vert_f[3]);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo2_data[1], (hipfftComplex*)geo2_data[1], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g4_data[1], geo2_data[1], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[1], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo2_data[1], d_x1[3], hori_row, hori_col, num_pts, d_hori_z[3], d_hori_f[3]);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[1], d_x2[3], vert_row, vert_col, num_pts, d_vert_z[3], d_vert_f[3]);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo2_data[1], (hipfftComplex*)geo2_data[1], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g3_data[1], geo2_data[1], N, true);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo1_data[1], (hipfftComplex*)geo1_data[1], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y3[3], geo1_data[1], hori_num, num_pts, N, d_hori_x[3], d_hori_b[3], false);

    // Far matrix multiplication for block Zme: FFT group 2
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[1], N);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g2_data[1], geo2_data[1], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[1], N);
    scatter<<< hori_col / 256 + 1, 256 >>>(geo2_data[1], d_x1[3], hori_row, hori_col, num_pts, d_hori_x[3], d_hori_f[3]);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo2_data[1], (hipfftComplex*)geo2_data[1], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g4_data[1], geo2_data[1], N, true);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo1_data[1], (hipfftComplex*)geo1_data[1], HIPFFT_BACKWARD);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y4[3], geo1_data[1], vert_num, num_pts, N, d_vert_y[3], d_vert_b[3], false);

    // Far matrix multiplication for block Zme: FFT group 3
    clearData<<< N / 256 + 1, 256 >>>(geo1_data[1], N);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g3_data[1], geo2_data[1], N, false);
    clearData<<< N / 256 + 1, 256 >>>(geo2_data[1], N);
    scatter<<< vert_col / 256 + 1, 256 >>>(geo2_data[1], d_x2[3], vert_row, vert_col, num_pts, d_vert_y[3], d_vert_f[3]);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo2_data[1], (hipfftComplex*)geo2_data[1], HIPFFT_FORWARD);
    convolveTransfer<<< N / 256 + 1, 256 >>>(geo1_data[1], g2_data[1], geo2_data[1], N, true);
    hipfftExecC2C(plan[3], (hipfftComplex*)geo1_data[1], (hipfftComplex*)geo1_data[1], HIPFFT_BACKWARD);
    accumulate<<< hori_num / 256 + 1, 256 >>>(d_y3[3], geo1_data[1], hori_num, num_pts, N, d_hori_z[3], d_hori_b[3], false);
    accumulate<<< vert_num / 256 + 1, 256 >>>(d_y4[3], geo1_data[1], vert_num, num_pts, N, d_vert_z[3], d_vert_b[3], false);
}

/// @brief Destroy the FFT computation plans and deallocated associated memory
void MVProd4::cleanAll() {
    hipsparseDestroy(handle0);
    hipsparseDestroy(handle1);
    hipsparseDestroy(handle2);
    hipsparseDestroy(handle3);
    for (int dev = 0; dev < 4; dev++) {
        hipSetDevice(devNumber[dev]);
        hipFree(d_Arows[dev]);
        hipFree(d_Acols[dev]);
        hipFree(d_Brows[dev]);
        hipFree(d_Bcols[dev]);
        hipFree(d_Aee[dev]);
        hipFree(d_Aem[dev]);
        hipFree(d_Amm[dev]);
        hipFree(d_Bee[dev]);
        hipFree(d_Bem[dev]);
        hipFree(d_Bmm[dev]);
        hipFree(d_x1[dev]);
        hipFree(d_x2[dev]);
        hipFree(d_x3[dev]);
        hipFree(d_x4[dev]);
        hipFree(d_y1[dev]);
        hipFree(d_y2[dev]);
        hipFree(d_y3[dev]);
        hipFree(d_y4[dev]);
        hipFree(d_base1[dev]);
        hipFree(d_base2[dev]);
        hipFree(zvals[dev]);
        hipFree(xvech[dev]);
        hipFree(wvech[dev]);
        hipFree(xvecl[dev]);
        hipFree(wvecl[dev]);
        hipFree(d_hori_x[dev]);
        hipFree(d_hori_z[dev]);
        hipFree(d_hori_d[dev]);
        hipFree(d_hori_f[dev]);
        hipFree(d_hori_b[dev]);
        hipFree(d_vert_y[dev]);
        hipFree(d_vert_z[dev]);
        hipFree(d_vert_d[dev]);
        hipFree(d_vert_f[dev]);
        hipFree(d_vert_b[dev]);
        hipFree(d_work[dev]);
        hipsparseDestroySpMat(Aee[dev]);
        hipsparseDestroySpMat(Aem[dev]);
        hipsparseDestroySpMat(Amm[dev]);
        hipsparseDestroySpMat(Bee[dev]);
        hipsparseDestroySpMat(Bem[dev]);
        hipsparseDestroySpMat(Bmm[dev]);
        hipsparseDestroyDnVec(x1[dev]);
        hipsparseDestroyDnVec(x2[dev]);
        hipsparseDestroyDnVec(x3[dev]);
        hipsparseDestroyDnVec(x4[dev]);
        hipsparseDestroyDnVec(y1[dev]);
        hipsparseDestroyDnVec(y2[dev]);
        hipsparseDestroyDnVec(y3[dev]);
        hipsparseDestroyDnVec(y4[dev]);
        hipfftDestroy(plan[dev]);
    }
    for (int count = 0; count < 2; count++) {
        hipSetDevice(devNumber[count]);
        hipFree(g0_data[count]);
        hipFree(g1_data[count]);
        hipFree(geo0_data[count]);
    }
    for (int count = 0; count < 2; count++) {
        hipSetDevice(devNumber[count + 2]);
        hipFree(g2_data[count]);
        hipFree(g3_data[count]);
        hipFree(g4_data[count]);
        hipFree(geo1_data[count]);
        hipFree(geo2_data[count]);
    }
}